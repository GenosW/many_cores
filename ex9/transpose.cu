#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "timer.hpp"

#define PRINT_ONLY 10


__global__
void transpose(double *A, int N)
{
  int t_idx = blockIdx.x*blockDim.x + threadIdx.x;
  int row_idx = t_idx / N;
  int col_idx = t_idx % N;
  
  if (row_idx < N && col_idx < N
    && col_idx > row_idx
    && t_idx < N*N) 
  {
    double tmp = A[row_idx * N + col_idx];
    A[row_idx * N + col_idx] = A[col_idx * N + row_idx];
    A[col_idx * N + row_idx] = tmp;
  }
}

__global__
void transpose_original(double *A, int N)
{
  int t_idx = blockIdx.x*blockDim.x + threadIdx.x;
  int row_idx = t_idx / N;
  int col_idx = t_idx % N;
  
  if (row_idx < N && col_idx < N) 
    A[row_idx * N + col_idx] = A[col_idx * N + row_idx];
}


void print_A(double *A, int N)
{
  int to_print = min(N, PRINT_ONLY);
  for (int i = 0; i < to_print; i++) {
    for (int j = 0; j < to_print; ++j) {
      std::cout << A[i * N + j] << ", ";
    }
    std::cout << std::endl;
  }
}

int main(void)
{
  int N = 100;

  double *A, *cuda_A;
  Timer timer;

  // Allocate host memory and initialize
  A = (double*)malloc(N*N*sizeof(double));
  
  for (int i = 0; i < N*N; i++) {
    A[i] = i;
  }

  print_A(A, N);


  // Allocate device memory and copy host data over
  hipMalloc(&cuda_A, N*N*sizeof(double)); 

  // copy data over
  hipMemcpy(cuda_A, A, N*N*sizeof(double), hipMemcpyHostToDevice);

  // wait for previous operations to finish, then start timings
  hipDeviceSynchronize();
  timer.reset();

  // Perform the transpose operation
  transpose<<<(N+255)/256, 256>>>(cuda_A, N);

  // wait for kernel to finish, then print elapsed time
  hipDeviceSynchronize();
  double elapsed = timer.get();
  std::cout << std::endl << "Time for transpose: " << elapsed << std::endl;
  std::cout << "Effective bandwidth: " << (2*N*N*sizeof(double)) / elapsed * 1e-9 << " GB/sec" << std::endl;
  std::cout << std::endl;

  // copy data back (implicit synchronization point)
  hipMemcpy(A, cuda_A, N*N*sizeof(double), hipMemcpyDeviceToHost);

  print_A(A, N);

  // My friend was a bit sloppy and forgot these two lines...
  free(A);
  hipFree(cuda_A);
  // Well, happens to the best!

  hipDeviceReset();  // for CUDA leak checker to work

  return EXIT_SUCCESS;
}

