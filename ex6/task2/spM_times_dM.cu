#include "hip/hip_runtime.h"
#include "timer.hpp"
#include "poisson2d.hpp"
#include <algorithm>
#include <numeric>
#include <cmath>
// #include <hipblas.h>
// #include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>

#define BLOCK_SIZE 256
#define GRID_SIZE 256
// #define SEP ";"
#define TESTS 5

// #define DEBUG
#ifndef DEBUG
  #define CSV
#endif

// START--------------- CONVENIENCE FUNTIONS ------------------START // 
// template <typename T>
// void printContainer(T* container, const int size) {
//   std::cout << *container;
//   for (int i = 1; i < size; ++i) 
//     std::cout << " | " << *(container+i) ;
//   std::cout << std::endl;
// }

template <typename T>
void printContainer(T container, const int size) {
  std::cout << container[0];
  for (int i = 1; i < size; ++i) 
    std::cout << " | " << container[i] ;
  std::cout << std::endl;
}

template <typename T>
void printContainer(T container, const int size, const int only) {
  std::cout << container[0];
  for (int i = 1; i < only; ++i) 
      std::cout  << " | " << container[i];
  std::cout << " | ...";
  for (int i = size - only; i < size; ++i) 
    std::cout  << " | " << container[i];
  std::cout << std::endl;
}

template <typename T>
void printContainerStrided(T container, const int size, const int stride) {
  std::cout << container[0];
  for (int i = stride; i < size; i+=stride) 
      std::cout  << " | " << container[i];
  std::cout << std::endl;
}

void printResults(double* results, std::vector<std::string> names, int size){
  std::cout << "Results:" << std::endl;
  for (int i = 0; i < size; ++i) {
    std::cout << names[i] << " : " << results[i] << std::endl;
  }
}

void printResults(double* results, double* ref, std::vector<std::string> names, int size){
  std::cout << "Results (with difference to reference):" << std::endl;
  for (int i = 0; i < size; ++i) {
    std::cout << names[i] << " = " << results[i] << " ||  " << ref[i] - results[i] << std::endl;
  }
}

template <typename T>
void toCSV(std::fstream& csv, T* array, int size) {
  csv << size;
  for (int i = 0; i < size; ++i) {
    csv << ";" << array[i];
  }
  csv << std::endl;
}

double median(std::vector<double>& vec)
{
  size_t size = vec.size();
  if (size == 0)
    return 0.;

  sort(vec.begin(), vec.end());
  size_t mid = size/2;
  return size % 2 == 0 ? (vec[mid] + vec[mid-1]) / 2 : vec[mid];
}
// END--------------- CONVENIENCE FUNCTIONS ------------------END // 

//
// START--------------- KERNELS ------------------START // 
//
// y = A * x
__global__ void cuda_csr_matvec_product(int N, int *csr_rowoffsets,
  int *csr_colindices, double *csr_values,
  double *x, double *y)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
    double sum = 0;
    for (int k = csr_rowoffsets[i]; k < csr_rowoffsets[i + 1]; k++) {
      sum += csr_values[k] * x[csr_colindices[k]];
    }
    y[i] = sum;
  }
}

// Y= A * X
__global__ void A_MatMul_Xrm(int N, int K,
  int *csr_rowoffsets, int *csr_colindices, double *csr_values,
  double *X, double *Y)
{
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {
    int row_start = csr_rowoffsets[tid];
    int row_end = csr_rowoffsets[tid + 1];

    // for (int k = 0; k < K; ++k){
    //   double sum = 0.0;
    //   for (int i = row_start; i < row_end; i++) {
    //     sum += csr_values[i]* X[csr_colindices[i]*K + k];
    //   }
    //   Y[k + tid*K] = sum;
    // }
    for (int k = 0; k < K; ++k){
      double sum = 0.0;
      for (int i = row_start; i < row_end; i++) {
        sum += csr_values[i]* X[csr_colindices[i]*K + k];
      }
      Y[k + tid*K] = sum;
    }

    // for (int i = row_start; i < row_end; i++) {
    //   double aij = csr_values[i];
    //   int row_of_X = csr_colindices[i]*K;
    //   for (int k = 0; k < K; ++k){
    //     Y[k + tid*K] += aij * X[row_of_X + k];
    //   }
    // }
  }
}

// Y= A * X
__global__ void A_MatMul_Xcm(int N, int K,
  int *csr_rowoffsets, int *csr_colindices, double *csr_values,
  double *X, double *Y)
{
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {
    int row_start = csr_rowoffsets[tid];
    int row_end = csr_rowoffsets[tid + 1];

    for (int k = 0; k < K; ++k){
      double sum = 0.0;
      for (int i = row_start; i < row_end; i++) {
        sum += csr_values[i]* X[csr_colindices[i] + k*N];
      }
      Y[k + tid*K] = sum;
    }
  }
}


// Y= A * X
__global__ void A_MatMul_Xrm_ik(int N, int K,
  int *csr_rowoffsets, int *csr_colindices, double *csr_values,
  double *X, double *Y)
{
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {
    int row_start = csr_rowoffsets[tid];
    int row_end = csr_rowoffsets[tid + 1];

    for (int i = row_start; i < row_end; i++) {
      double aij = csr_values[i];
      int colindex = csr_colindices[i];
      for (int k = 0; k < K; ++k){
        Y[k + tid*K] += aij * X[colindex*K + k];
      }
    }
  }
}

// Y= A * X
__global__ void A_MatMul_Xcm_ik(int N, int K,
  int *csr_rowoffsets, int *csr_colindices, double *csr_values,
  double *X, double *Y)
{
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {
    int row_start = csr_rowoffsets[tid];
    int row_end = csr_rowoffsets[tid + 1];

    for (int i = row_start; i < row_end; i++) {
      double aij = csr_values[i];
      int colindex = csr_colindices[i];
      for (int k = 0; k < K; ++k){
        Y[k + tid*K] += aij* X[colindex + k*N];
      }
    }
  }
}

// // Y = A * X
// __global__ void A_MatMul_Xrm(int N, int K, int *csr_rowoffsets,
//   int *csr_colindices, double *csr_values,
//   double *x, double *y)
// {
//   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {

//     for (int k = 0; k < K; k++) {
//       double sum = 0;

//       for (int jj = csr_rowoffsets[i]; jj < csr_rowoffsets[i + 1]; jj++) {
//         sum += csr_values[jj] * x[csr_colindices[jj]*K + k];
//       }

//       y[i + N*k] = sum;
//     }

//   }
// }


// // Y = A * X
// __global__ void A_MatMul_Xcm(int N, int K, int *csr_rowoffsets,
//   int *csr_colindices, double *csr_values,
//   double *x, double *y)
// {
//   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {

//     for (int k = 0; k < K; k++) {
//       double sum = 0;

//       for (int jj = csr_rowoffsets[i]; jj < csr_rowoffsets[i + 1]; jj++) {
//         sum += csr_values[jj] * x[csr_colindices[jj] + N*k];
//       }

//       y[i + N*k] = sum;
//     }

//   }
// }
// END--------------- KERNELS ------------------END // 
//

int main(void) {
  Timer timer;
  std::vector<int> vec_Ns{100, 1024, 10000, 100489, 1000000};
  // std::vector<int> vec_Ns{1000000};
  std::vector<int> vec_Ks{2, 4, 8, 16};
  // std::vector<int> vec_Ks{3, 5, 9, 15};
  std::vector<double> times(TESTS, 0);

#ifdef CSV
  std::fstream csv_times;
  std::string csv_times_name = "ph_data_ik.csv";
  csv_times.open(csv_times_name, std::fstream::out | std::fstream::trunc);

  std::string header = "N;K;time_single;time_rm_stacked;time_cm_stacked";
    // to csv file
  csv_times << header << std::endl;
#endif

  for (int& N : vec_Ns) {
    for (int& K : vec_Ks) {
      // hipblasHandle_t h;
      // hipblasCreate(&h);

      //
      // allocate + init host memory:
      //
  #ifdef DEBUG
      std::cout << "Allocating host + device arrays..." << std::endl;
  #endif
      // "Vectors"
      double* X = (double *)malloc(sizeof(double) * N * K);
      double* Y = (double *)malloc(sizeof(double) * N * K);
      double* Y2 = (double *)malloc(sizeof(double) * N * K);
      // double* x = (double *)malloc(sizeof(double) * N);
      double* y = (double *)malloc(sizeof(double) * N);
      std::fill(X, X + (N*K), 1.);
      std::fill(Y, Y + (N*K), 0.);
      std::fill(Y2, Y2 + (N*K), 0.);
      // std::fill(x, x + N, 1.);

      double *cuda_X;
      double *cuda_Y;
      // double *cuda_Y2;
      // double *cuda_x;
      double *cuda_y;
      hipMalloc(&cuda_X, sizeof(double) * N*K);
      hipMalloc(&cuda_Y, sizeof(double) * N*K);
      // hipMalloc(&cuda_Y2, sizeof(double) * N*K);
      // hipMalloc(&cuda_x, sizeof(double) * N);
      hipMalloc(&cuda_y, sizeof(double) * N);

      // Matrix
      int* csr_rowoffsets = (int* )malloc(sizeof(int) * (N+1));
      int* csr_colindices = (int* )malloc(sizeof(int) * 5*N);
      double* csr_values = (double* )malloc(sizeof(double) * 5*N);

      int* cuda_csr_rowoffsets; 
      int* cuda_csr_colindices;
      double* cuda_csr_values;
      hipMalloc(&cuda_csr_rowoffsets, sizeof(int) * (N+1));
      hipMalloc(&cuda_csr_colindices, sizeof(int) * 5*N);
      hipMalloc(&cuda_csr_values, sizeof(double) * 5*N);
      //
      // Copy data to GPU
      //
  #ifdef DEBUG
      std::cout << "Copying data to GPU..." << std::endl;
  #endif
      hipMemcpy(cuda_X, X, sizeof(double) * N*K, hipMemcpyHostToDevice);
      hipMemcpy(cuda_Y, Y, sizeof(double) * N*K, hipMemcpyHostToDevice);
      // hipMemcpy(cuda_Y2, Y2, sizeof(double) * N*K, hipMemcpyHostToDevice);
      // hipMemcpy(cuda_x, X, sizeof(double) * N, hipMemcpyHostToDevice);
  //    hipMemcpy(cuda_y, y, sizeof(double) * N*K, hipMemcpyHostToDevice);

  // Assemble A
  #ifdef DEBUG
      std::cout << "Generating A..." << std::endl;
  #endif
      generate_fdm_laplace(sqrt(N), csr_rowoffsets, csr_colindices, csr_values);
  #ifdef DEBUG
      std::cout << "Generating A done!" << std::endl;
  #endif
      hipMemcpy(cuda_csr_rowoffsets, csr_rowoffsets, sizeof(int) * (N+1), hipMemcpyHostToDevice);
      hipMemcpy(cuda_csr_colindices, csr_colindices, sizeof(int) * 5*N, hipMemcpyHostToDevice);
      hipMemcpy(cuda_csr_values, csr_values, sizeof(double) * 5*N, hipMemcpyHostToDevice);  
      
      // ------------------ TEST ---------------- //

#ifdef DEBUG
      std::cout << "N = " << N << std::endl;
      std::cout << "K = " << K << std::endl;

      std::cout << "Running per vector product kernel K times..." << std::endl;
#endif
      for (int iter = 0; iter < TESTS; iter++){
        timer.reset();
        for (int k = 0; k < K; ++k)
          cuda_csr_matvec_product<<<GRID_SIZE, BLOCK_SIZE>>>(
            N, 
            cuda_csr_rowoffsets, cuda_csr_colindices, cuda_csr_values,
            cuda_X, cuda_y);
        hipMemcpy(y, cuda_y, sizeof(double) * N, hipMemcpyDeviceToHost);
        times[iter] = timer.get();
      } 
      double time_single = median(times);

  #ifdef DEBUG
      std::cout << "Running RowMajor stacked kernel..." << std::endl;
  #endif
      for (int iter = 0; iter < TESTS; iter++){
        timer.reset();
        A_MatMul_Xrm_ik<<<GRID_SIZE, BLOCK_SIZE>>>(
            N, K,
            cuda_csr_rowoffsets, cuda_csr_colindices, cuda_csr_values,
            cuda_X, cuda_Y);
        hipMemcpy(Y, cuda_Y, sizeof(double) * N*K, hipMemcpyDeviceToHost);
        times[iter] = timer.get();
      }
      double time_rm_stacked = median(times);

  #ifdef DEBUG
      std::cout << "Running ColumnMajor stacked kernel..." << std::endl;
  #endif
      hipMemcpy(cuda_Y, Y2, sizeof(double) * N*K, hipMemcpyHostToDevice);
      for (int iter = 0; iter < TESTS; iter++){
        timer.reset();
        A_MatMul_Xcm_ik<<<GRID_SIZE, BLOCK_SIZE>>>(
            N, K,
            cuda_csr_rowoffsets, cuda_csr_colindices, cuda_csr_values,
            cuda_X, cuda_Y);
        hipMemcpy(Y2, cuda_Y, sizeof(double) * N*K, hipMemcpyDeviceToHost);
        times[iter] = timer.get();
      }
      double time_cm_stacked = median(times);




      //
      // Compare results
      //
  #ifdef DEBUG
      std::cout << "DEBUG output:" << std::endl;
      // int only = 4;
      std::cout << "A (non zero entries by row)" << std::endl;
      // int csr_values_size = csr_rowoffsets[N+1];
      // printContainer(y, N);
      std::cout << "Row" << std::endl;
      int max_output = 10;
      for (int row = 0; row < min(N, max_output); row++){
        std::cout << row << ": ";
        printContainer(csr_values + csr_rowoffsets[row], min(csr_rowoffsets[row+1]-csr_rowoffsets[row], max_output));
      }

      std::cout << "y:" << std::endl;
      printContainer(y, min(N, max_output));
      std::cout << "Y_rm:" << std::endl;
      printContainerStrided(Y, min(N, max_output)*K, K);
      std::cout << "Y_cm:" << std::endl;
      printContainerStrided(Y2, min(N, max_output)*K, K);


      std::cout << "Single runtime: " << time_single << std::endl;
      std::cout << "RM Stacked runtime: " << time_rm_stacked << std::endl;
      std::cout << "CM Stacked runtime: " << time_cm_stacked << std::endl;

      //
      // Clean up:
      //
      std::cout << "----------------------------------------------------" << std::endl;
      std::cout << "Cleaning up..." << std::endl;
  #endif

  #ifdef CSV
      std::string sep = ";";
      csv_times << N << sep
                << K << sep
                << time_single << sep
                << time_rm_stacked << sep
                << time_cm_stacked 
                << std::endl;
  #endif
    free(X);
    free(Y);
    free(Y2);
    // free(x);
    free(y);
    free(csr_rowoffsets); 
    free(csr_colindices);
    free(csr_values);

    hipFree(cuda_X);
    hipFree(cuda_Y);
    // hipFree(cuda_Y2);
    // hipFree(cuda_x);
    hipFree(cuda_y);
    hipFree(cuda_csr_rowoffsets); 
    hipFree(cuda_csr_colindices);
    hipFree(cuda_csr_values);
#ifdef DEBUG
    std::cout << "Clean up done!" << std::endl;
#endif
    }
  }

#ifdef CSV
  csv_times.close();
  
  std::cout << "\nRuntimes in csv form can be found here\nhttps://gtx1080.360252.org/2020/ex6/" + csv_times_name << std::endl;
#endif
  return EXIT_SUCCESS;
}
