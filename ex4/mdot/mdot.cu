#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include "timer.hpp"

#define BLOCK_SIZE 256
#define GRID_SIZE 256

__global__ void xDOTy(const size_t N, double* x, 
  double* y,
  double* z)
{
  size_t tid = threadIdx.x + blockDim.x* blockIdx.x;
  const size_t stride = blockDim.x* gridDim.x;

  __shared__ double cache[BLOCK_SIZE];

  double tid_sum = 0.0;
  for (; tid < N; tid += stride)
  {
    double tmp_x = x[tid];
    tid_sum +=  tmp_x * y[tid];
  }
  tid = threadIdx.x;
  cache[tid] = tid_sum;

  __syncthreads();
  for (size_t i = blockDim.x/2; i != 0; i /=2)
  {
    __syncthreads();
    if (tid < i) //lower half does smth, rest idles
      cache[tid] += cache[tid + i]; //lower looks up by stride and sums up
  }

  if(tid == 0) // cache[0] now contains block_sum
  {
    atomicAdd(z, cache[0]);
  }
}

/** Computes 8 vector dot products of type <x,y_i> with i=1,...,8 at once

z should be a pointer to an array of size 8 to store the results.
*/
__global__ void xDOTy8(const size_t N, double* x, 
                      double* y1, double* y2,
                      double* y3, double* y4,
                      double* y5, double* y6,
                      double* y7, double* y8,
                      double* z)
{
  size_t tid = threadIdx.x + blockDim.x* blockIdx.x;
  const size_t stride = blockDim.x* gridDim.x;

  __shared__ double cache[8][BLOCK_SIZE];

  double tid_sum1 = 0.0; double tid_sum2 = 0.0;
  double tid_sum3 = 0.0; double tid_sum4 = 0.0;
  double tid_sum5 = 0.0; double tid_sum6 = 0.0;
  double tid_sum7 = 0.0; double tid_sum8 = 0.0;
  for (; tid < N; tid += stride)
  {
    double tmp_x = x[tid];
    tid_sum1 +=  tmp_x * y1[tid];
    tid_sum2 +=  tmp_x * y2[tid];
    tid_sum3 +=  tmp_x * y3[tid];
    tid_sum4 +=  tmp_x * y4[tid];
    tid_sum5 +=  tmp_x * y5[tid];
    tid_sum6 +=  tmp_x * y6[tid];
    tid_sum7 +=  tmp_x * y7[tid];
    tid_sum8 +=  tmp_x * y8[tid];
  }
  tid = threadIdx.x;
  cache[0][tid] = tid_sum1; cache[1][tid] = tid_sum2;
  cache[2][tid] = tid_sum3; cache[3][tid] = tid_sum4;
  cache[4][tid] = tid_sum5; cache[5][tid] = tid_sum6;
  cache[6][tid] = tid_sum7; cache[7][tid] = tid_sum8;

  __syncthreads();
  for (size_t i = blockDim.x/2; i != 0; i /=2)
  {
    __syncthreads();
    if (tid < i) { //lower half
      cache[0][tid] += cache[0][tid + i];
      cache[1][tid] += cache[1][tid + i];
      cache[2][tid] += cache[2][tid + i];
      cache[4][tid] += cache[4][tid + i];
    // }
    // else if (tid < i*2){
      cache[5][tid] += cache[5][tid + i];
      cache[6][tid] += cache[6][tid + i];
      cache[7][tid] += cache[7][tid + i];
      cache[8][tid] += cache[8][tid + i];
    }

  }

  if (tid==0) // cache[0] now contains block_sum
  {
    for (int i = 0; i < 8; ++i)
    atomicAdd(z+i, cache[i][0]);
  }
}

int main(void)
{
    const size_t N = 100000;
    const size_t K = 8;
    Timer timer;

    //
    // Initialize CUBLAS:
    //
    std::cout << "Init CUBLAS..." << std::endl;
    hipblasHandle_t h;
    hipblasCreate(&h);


    //
    // allocate host memory:
    //
    std::cout << "Allocating host arrays..." << std::endl;
    double  *x = (double*)malloc(sizeof(double) * N);
    double **y = (double**)malloc(sizeof(double*) * K);
    for (size_t i=0; i<K; ++i) {
      y[i] = (double*)malloc(sizeof(double) * N);
    }
    double *results  = (double*)malloc(sizeof(double) * K);
    double *results2 = (double*)malloc(sizeof(double) * K);
    double *results_xDOTy = (double*)malloc(sizeof(double) * K);
    double *results_xDOTy8 = (double*)malloc(sizeof(double) * K);
    std::fill(results_xDOTy, results_xDOTy+K, 0.0);
    std::fill(results_xDOTy8, results_xDOTy8+K, 0.0);


    //
    // allocate device memory
    //
    std::cout << "Allocating CUDA arrays..." << std::endl;
    double *cuda_x; hipMalloc(&cuda_x, sizeof(double)*N);
    double **cuda_y = (double**)malloc(sizeof(double*) * K);  // storing CUDA 
    double *cuda_results; hipMalloc(&cuda_results, sizeof(double)*K);
    for (size_t i=0; i<K; ++i) {
      hipMalloc( (void **)(&cuda_y[i]), sizeof(double)*N);
    }

    //
    // fill host arrays with values
    //
    for (size_t j=0; j<N; ++j) {
      x[j] = 1 + j%K;
    }
    for (size_t i=0; i<K; ++i) {
      for (size_t j=0; j<N; ++j) {
        y[i][j] = 1 + rand() / (1.1 * RAND_MAX);
      }
    }

    //
    // Reference calculation on CPU:
    //
    timer.reset();
    for (size_t i=0; i<K; ++i) {
      results[i] = 0;
      results2[i] = 0;
      for (size_t j=0; j<N; ++j) {
        results[i] += x[j] * y[i][j];
      }
    }    
    double time_CPU = timer.get();
   
    //
    // Copy data to GPU
    //
    std::cout << "Copying data to GPU..." << std::endl;
    hipMemcpy(cuda_x, x, sizeof(double)*N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_results, results_xDOTy8, sizeof(double)*K, hipMemcpyHostToDevice);
    for (size_t i=0; i<K; ++i) {
      hipMemcpy(cuda_y[i], y[i], sizeof(double)*N, hipMemcpyHostToDevice);
    }


    //
    // Let CUBLAS do the work:
    //
    std::cout << "Running dot products with CUBLAS..." << std::endl;
    timer.reset();
    for (size_t i=0; i<K; ++i) {
      hipblasDdot(h, N, cuda_x, 1, cuda_y[i], 1, results2 + i);
    }
    double time_cublas = timer.get();

    //
    // Let xDOTy do the work:
    //
    std::cout << "Running dot products with custom xDOTy8..." << std::endl;
    timer.reset();
    for (size_t i=0; i<K; ++i) {
      xDOTy<<<GRID_SIZE, BLOCK_SIZE>>>(N, 
            cuda_x, cuda_y[i],
            cuda_results+i);
    }
    hipMemcpy(results_xDOTy, cuda_results, sizeof(double)*K, hipMemcpyDeviceToHost);
    double time_xDOTy = timer.get();

    //
    // Let xDOTy8 do the work:
    //
    hipMemcpy(cuda_results, results_xDOTy8, sizeof(double)*K, hipMemcpyHostToDevice);
    std::cout << "Running dot products with custom xDOTy8..." << std::endl;
    timer.reset();
    for (size_t i=0; i<(int)K/8; ++i) {
      int batch_offset = (i*8);
      xDOTy8<<<GRID_SIZE, BLOCK_SIZE>>>(N, 
            cuda_x, 
            cuda_y[batch_offset], cuda_y[batch_offset+1],
            cuda_y[batch_offset+2], cuda_y[batch_offset+3],
            cuda_y[batch_offset+4], cuda_y[batch_offset+5],
            cuda_y[batch_offset+6], cuda_y[batch_offset+7],
            cuda_results+batch_offset);
    }
    hipMemcpy(results_xDOTy8, cuda_results, sizeof(double)*K, hipMemcpyDeviceToHost);
    double time_xDOTy8 = timer.get();

    //
    // Compare results
    //
    std::cout << "------------------------------------------------------------" << std::endl;
    std::cout << "Copying results back to host..." << std::endl;
    for (size_t i=0; i<K; ++i) {
      std::cout << results[i] << " on CPU, " << results2[i] << " on GPU. Relative difference: " << fabs(results[i] - results2[i]) / results[i] << std::endl;
    }std::cout << "------------------------------------------------------------" << std::endl;
    std::cout << "Now to compare the custom kernel xDOTy to CPU..." << std::endl;
    for (size_t i=0; i<K; ++i) {
      std::cout << results[i] << " on CPU, " << results_xDOTy[i] << " on GPU. Relative difference: " << fabs(results[i] - results_xDOTy[i]) / results[i] << std::endl;
    }
    std::cout << "------------------------------------------------------------" << std::endl;
    std::cout << "Now to compare the custom kernel xDOTy8 to CPU..." << std::endl;
    for (size_t i=0; i<K; ++i) {
      std::cout << results[i] << " on CPU, " << results_xDOTy8[i] << " on GPU. Relative difference: " << fabs(results[i] - results_xDOTy8[i]) / results[i] << std::endl;
    }

    std::cout << "------------------------------------------------------------" << std::endl;
    std::cout << "And now compare the runtime of all implementations..." << std::endl;
    std::cout << "CPU......" << time_CPU << "s >> Speedup: " << 100-time_CPU/time_CPU*100 << "%" << std::endl;
    std::cout << "CUBAS...." << time_cublas << "s >> Speedup: " << 100-time_cublas/time_CPU*100 << "%" << std::endl;
    std::cout << "xDOTy..." << time_xDOTy << "s >> Speedup: " << 100-time_xDOTy/time_CPU*100 << "%" << std::endl;
    std::cout << "xDOTy8..." << time_xDOTy8 << "s >> Speedup: " << 100-time_xDOTy8/time_CPU*100 << "%" << std::endl;

    
    
    //
    // Clean up:
    //
    std::cout << "Cleaning up..." << std::endl;
    free(x);
    hipFree(cuda_x);

    for (size_t i=0; i<K; ++i) {
      free(y[i]);
      hipFree(cuda_y[i]);
    }
    free(y);
    free(cuda_y);

    free(results);
    free(results2);
    free(results_xDOTy8);
 
    hipblasDestroy(h);
    return 0;
}
