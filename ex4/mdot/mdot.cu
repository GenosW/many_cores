#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include "timer.hpp"

#define BLOCK_SIZE 256
#define GRID_SIZE 256

__global__ void xDOTy(const size_t N, double* x, 
  double* y,
  double* z)
{
  size_t tid = threadIdx.x + blockDim.x* blockIdx.x;
  const size_t stride = blockDim.x* gridDim.x;

  __shared__ double cache[BLOCK_SIZE];

  double tid_sum = 0.0;
  for (; tid < N; tid += stride)
  {
    double tmp_x = x[tid];
    tid_sum +=  tmp_x * y[tid];
  }
  tid = threadIdx.x;
  cache[tid] = tid_sum;

  __syncthreads();
  for (size_t i = blockDim.x/2; i != 0; i /=2)
  {
    __syncthreads();
    if (tid < i) //lower half does smth, rest idles
      cache[tid] += cache[tid + i]; //lower looks up by stride and sums up
  }

  if(tid == 0) // cache[0] now contains block_sum
  {
    atomicAdd(z, cache[0]);
  }
}

/** Computes 8 vector dot products of type <x,y_i> with i=1,...,8 at once

z should be a pointer to an array of size 8 to store the results.
*/
__global__ void xDOTy8(const size_t N, double* x, 
                      double* y1, double* y2,
                      double* y3, double* y4,
                      double* y5, double* y6,
                      double* y7, double* y8,
                      double* z)
{
  size_t tid = threadIdx.x + blockDim.x* blockIdx.x;
  const size_t stride = blockDim.x* gridDim.x;

  __shared__ double cache1[BLOCK_SIZE];
  __shared__ double cache2[BLOCK_SIZE];
  __shared__ double cache3[BLOCK_SIZE];
  __shared__ double cache4[BLOCK_SIZE];
  __shared__ double cache5[BLOCK_SIZE];
  __shared__ double cache6[BLOCK_SIZE];
  __shared__ double cache7[BLOCK_SIZE];
  __shared__ double cache8[BLOCK_SIZE];

  double tid_sum1 = 0.0; double tid_sum2 = 0.0;
  double tid_sum3 = 0.0; double tid_sum4 = 0.0;
  double tid_sum5 = 0.0; double tid_sum6 = 0.0;
  double tid_sum7 = 0.0; double tid_sum8 = 0.0;
  for (; tid < N; tid += stride)
  {
    double tmp_x = x[tid];
    tid_sum1 +=  tmp_x * y1[tid];
    tid_sum2 +=  tmp_x * y2[tid];
    tid_sum3 +=  tmp_x * y3[tid];
    tid_sum4 +=  tmp_x * y4[tid];
    tid_sum5 +=  tmp_x * y5[tid];
    tid_sum6 +=  tmp_x * y6[tid];
    tid_sum7 +=  tmp_x * y7[tid];
    tid_sum8 +=  tmp_x * y8[tid];
  }
  tid = threadIdx.x;
  cache1[tid] = tid_sum1; 
  cache2[tid] = tid_sum2;
  cache3[tid] = tid_sum3; 
  cache4[tid] = tid_sum4;
  cache5[tid] = tid_sum5; 
  cache6[tid] = tid_sum6;
  cache7[tid] = tid_sum7; 
  cache8[tid] = tid_sum8;

  __syncthreads();
  for (size_t i = blockDim.x/2; i != 0; i /=2)
  {
    __syncthreads();
    if (tid < i) { //lower half
      cache1[tid] += cache1[tid + i];
      cache2[tid] += cache2[tid + i];
      cache3[tid] += cache3[tid + i];
      cache4[tid] += cache4[tid + i];
    // }
    // else if (tid < i*2){
      cache5[tid] += cache5[tid + i];
      cache6[tid] += cache6[tid + i];
      cache7[tid] += cache7[tid + i];
      cache8[tid] += cache8[tid + i];
    }

  }

  if (tid==0) // cache0 now contains block_sum
  {
    atomicAdd(z, cache1[0]);
    atomicAdd(z+1, cache2[0]);
    atomicAdd(z+2, cache3[0]);
    atomicAdd(z+3, cache4[0]);
    atomicAdd(z+4, cache5[0]);
    atomicAdd(z+5, cache6[0]);
    atomicAdd(z+6, cache7[0]);
    atomicAdd(z+7, cache8[0]);
  }
}

int main(void)
{
  const size_t K = 32;
  Timer timer;

  std::vector<double> times_CPU; 
  std::vector<double> times_cublas; 
  std::vector<double> times_xDOTy; 
  std::vector<double> times_xDOTy8;
  std::vector<int> vec_ks;
  std::vector<int> vec_Ns;

  std::fstream csv;
  csv.open ("ph_data.txt", std::fstream::out | std::fstream::trunc);

  for (size_t N = 1000; N <= 1000000; N*=10) {
    //
    // Initialize CUBLAS:
    //
    std::cout << "Init CUBLAS..." << std::endl;
    hipblasHandle_t h;
    hipblasCreate(&h);

    //
    // allocate host memory:
    //
    std::cout << "Allocating host arrays..." << std::endl;
    double  *x = (double*)malloc(sizeof(double) * N);
    double **y = (double**)malloc(sizeof(double*) * K);
    for (size_t i=0; i<K; ++i) {
      y[i] = (double*)malloc(sizeof(double) * N);
    }
    double *results  = (double*)malloc(sizeof(double) * K);
    double *results2 = (double*)malloc(sizeof(double) * K);
    double *results_xDOTy = (double*)malloc(sizeof(double) * K);
    double *results_xDOTy8 = (double*)malloc(sizeof(double) * K);
    std::fill(results_xDOTy, results_xDOTy+K, 0.0);
    std::fill(results_xDOTy8, results_xDOTy8+K, 0.0);


    //
    // allocate device memory
    //
    std::cout << "Allocating CUDA arrays..." << std::endl;
    double *cuda_x; hipMalloc(&cuda_x, sizeof(double)*N);
    double **cuda_y = (double**)malloc(sizeof(double*) * K);  // storing CUDA 
    double *cuda_results; hipMalloc(&cuda_results, sizeof(double)*K);
    for (size_t i=0; i<K; ++i) {
      hipMalloc( (void **)(&cuda_y[i]), sizeof(double)*N);
    }

    //
    // fill host arrays with values
    //
    for (size_t j=0; j<N; ++j) {
      x[j] = 1 + j%K;
    }
    for (size_t i=0; i<K; ++i) {
      for (size_t j=0; j<N; ++j) {
        y[i][j] = 1 + rand() / (1.1 * RAND_MAX);
      }
      results2[i] = 0;
    } 

    //
    // Copy data to GPU
    //
    std::cout << "Copying data to GPU..." << std::endl;
    hipMemcpy(cuda_x, x, sizeof(double)*N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_results, results_xDOTy8, sizeof(double)*K, hipMemcpyHostToDevice);
    for (size_t i=0; i<K; ++i) {
      hipMemcpy(cuda_y[i], y[i], sizeof(double)*N, hipMemcpyHostToDevice);
    }

    for (int k = 8; k <= K; k+=8) {
      
      //
      // Reference calculation on CPU:
      //
      timer.reset();
      for (size_t i=0; i<k; ++i) {
        results[i] = 0;
        for (size_t j=0; j<N; ++j) {
          results[i] += x[j] * y[i][j];
        }
      }    
      double time_CPU = timer.get();

      //
      // Let CUBLAS do the work:
      //
      std::cout << "Running dot products with CUBLAS..." << std::endl;
      timer.reset();
      for (size_t i=0; i<k; ++i) {
        hipblasDdot(h, N, cuda_x, 1, cuda_y[i], 1, results2 + i);
      }
      double time_cublas = timer.get();

      //
      // Let xDOTy do the work:
      //
      std::cout << "Running dot products with custom xDOTy8..." << std::endl;
      timer.reset();
      for (size_t i=0; i<k; ++i) {
        xDOTy<<<GRID_SIZE, BLOCK_SIZE>>>(N, 
              cuda_x, cuda_y[i],
              cuda_results+i);
      }
      hipMemcpy(results_xDOTy, cuda_results, sizeof(double)*k, hipMemcpyDeviceToHost);
      double time_xDOTy = timer.get();

      //
      // Let xDOTy8 do the work:
      //
      hipMemcpy(cuda_results, results_xDOTy8, sizeof(double)*k, hipMemcpyHostToDevice);
      std::cout << "Running dot products with custom xDOTy8..." << std::endl;
      timer.reset();
      for (size_t i=0; i<(int)k/8; ++i) {
        int batch_offset = (i*8);
        xDOTy8<<<GRID_SIZE, BLOCK_SIZE>>>(N, 
              cuda_x, 
              cuda_y[batch_offset], cuda_y[batch_offset+1],
              cuda_y[batch_offset+2], cuda_y[batch_offset+3],
              cuda_y[batch_offset+4], cuda_y[batch_offset+5],
              cuda_y[batch_offset+6], cuda_y[batch_offset+7],
              cuda_results+batch_offset);
      }
      hipMemcpy(results_xDOTy8, cuda_results, sizeof(double)*k, hipMemcpyDeviceToHost);
      double time_xDOTy8 = timer.get();

      //
      // Compare results
      //
      if (k==8) {
        std::cout << "------------------------------------------------------------" << std::endl;
        std::cout << "Copying results back to host..." << std::endl;
        for (size_t i=0; i<k; ++i) {
          std::cout << results[i] << " on CPU, " << results2[i] << " on GPU. Relative difference: " << fabs(results[i] - results2[i]) / results[i] << std::endl;
        }std::cout << "------------------------------------------------------------" << std::endl;
        std::cout << "Now to compare the custom kernel xDOTy to CPU..." << std::endl;
        for (size_t i=0; i<k; ++i) {
          std::cout << results[i] << " on CPU, " << results_xDOTy[i] << " on GPU. Relative difference: " << fabs(results[i] - results_xDOTy[i]) / results[i] << std::endl;
        }
        std::cout << "------------------------------------------------------------" << std::endl;
        std::cout << "Now to compare the custom kernel xDOTy8 to CPU..." << std::endl;
        for (size_t i=0; i<k; ++i) {
          std::cout << results[i] << " on CPU, " << results_xDOTy8[i] << " on GPU. Relative difference: " << fabs(results[i] - results_xDOTy8[i]) / results[i] << std::endl;
        }
      }

      bool in_percent = false;
      auto speedup = [ref_time=time_CPU, in_percent] (double comp_time) -> double { return (in_percent) ? (ref_time/comp_time)*100 : ref_time/comp_time;};
      auto time_in_ms = [] (double time) -> double { return time*1e-3;};
      std::cout << "------------------------------------------------------------" << std::endl;
      std::cout << "And now compare the runtime of all implementations..." << std::endl;
      std::string s_unit = (in_percent) ? "%" : "";
      std::string t_unit = "ms";
      std::cout << "CPU.........." << time_in_ms(time_CPU) << t_unit << std::endl;
      std::cout << "CUBLAS..." << time_in_ms(time_cublas) << t_unit << " >> Speedup: " << speedup(time_cublas) << s_unit << std::endl;
      std::cout << "xDOTy......" << time_in_ms(time_xDOTy) << t_unit << " >> Speedup: " << speedup(time_xDOTy) << s_unit << std::endl;
      std::cout << "xDOTy8...." << time_in_ms(time_xDOTy8) << t_unit << " >> Speedup: " << speedup(time_xDOTy8) << s_unit << std::endl;

      times_CPU.push_back(time_CPU); 
      times_cublas.push_back(time_cublas); 
      times_xDOTy.push_back(time_xDOTy); 
      times_xDOTy8.push_back(time_xDOTy8);
      vec_ks.push_back(k);
      vec_Ns.push_back(N);
    }
    
    
    //
    // Clean up:
    //
    std::cout << "Cleaning up..." << std::endl;
    free(x);
    hipFree(cuda_x);

    for (size_t i=0; i<K; ++i) {
      free(y[i]);
      hipFree(cuda_y[i]);
    }
    free(y);
    free(cuda_y);

    free(results);
    free(results2);
    free(results_xDOTy8);

    hipblasDestroy(h);
  }

  std::cout << "--------------------------CSV--------------------------------" << std::endl;
  std::string sep = ";";
  // to std::out
  std::cout << "N" << sep << "k" << sep << "time_CPU" << sep << "time_cublas" << sep << "time_xDOTy" << sep << "time_xDOTy8\n";
  for (int i = 0; i < vec_ks.size(); ++i ) {
    std::cout << std::scientific << vec_Ns[i] << sep 
      << vec_ks[i] << sep 
      << times_CPU[i] << sep 
      << times_cublas[i] << sep 
      << times_xDOTy[i] << sep 
      << times_xDOTy8[i] << "\n";
  }
  std::cout << std::endl;
  // to csv file
  csv << "N" << sep << "k" << sep << "time_CPU" << sep << "time_cublas" << sep << "time_xDOTy" << sep << "time_xDOTy8\n";
  for (int i = 0; i < vec_ks.size(); ++i ) {
    csv << std::scientific << vec_Ns[i] << sep 
      << vec_ks[i] << sep 
      << times_CPU[i] << sep 
      << times_cublas[i] << sep 
      << times_xDOTy[i] << sep 
      << times_xDOTy8[i] << "\n";
  }
  csv << std::endl;
  csv.close();

  return 0;
}
