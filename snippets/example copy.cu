
#include <hip/hip_runtime.h>
#include <iostream>


int main(void)
{

  double  *x, *y, *d_x;

  // Allocate host memory and initialize
  x = (double*)malloc(sizeof(double ));
  *x = 0.5;
  y = (double*)malloc(sizeof(double));
  *y = 1.5;
  // Allocate device memory and copy host data over
  hipMalloc(&d_x, sizeof(double)); 

  hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_x, y, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(x, d_x, sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  std::cout << *x << std::endl;


  hipFree(d_x);
  free(x);
  free(y);

  return EXIT_SUCCESS;
}

